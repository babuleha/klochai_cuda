#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipblas.h>
#include <hiprand.h>
#include <math.h>

#define THREADS_PER_BLOCK 32

// Функция для генерации случайной матрицы
void random_doubles(double* d_a, int size) {
    static hiprandGenerator_t gen = NULL;
    if (gen == NULL) {
        hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
        hiprandSetPseudoRandomGeneratorSeed(gen, (unsigned long long)time(NULL));
    }
    hiprandGenerateUniformDouble(gen, d_a, size);
}

// Кернел для симметризации матрицы
__global__ void gen_symm(double *d_a, double *d_tmp, int n) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < n && y < n) { 
        d_a[y * n + x] = d_tmp[y * n + x] + d_tmp[x * n + y];
    }
}

// Функция для генерации положительно определённой матрицы
void gen_dpm(double *d_a, double *d_tmp, int n) {
    double alpha = 1.0;
    double beta = 0.0;
    static hipblasHandle_t handle = NULL;
    if (handle == NULL) {
        hipblasCreate(&handle);
    }
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_tmp, n, d_tmp, n, &beta, d_a, n);
}

// Генерация матриц A и B
void gen_matrix(double *d_a, double *d_b, int n) {
    double *d_tmp;
    hipMalloc((void **)&d_tmp, n * n * sizeof(double));

    dim3 gridDim((n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
    dim3 blockDim(THREADS_PER_BLOCK, THREADS_PER_BLOCK);

    random_doubles(d_tmp, n * n);
    gen_symm<<<gridDim, blockDim>>>(d_a, d_tmp, n);
    hipDeviceSynchronize();

    random_doubles(d_tmp, n * n);
    gen_symm<<<gridDim, blockDim>>>(d_tmp, d_b, n);
    hipDeviceSynchronize();
    gen_dpm(d_b, d_tmp, n);

    hipFree(d_tmp);
}

// Диагонализация на GPU
void eig_gpu(hipsolverHandle_t cusolverH, double *d_a, double *d_b, double *d_eig, int n) {
    int *d_info;
    double *work;
    int lwork = 0;

    hipMalloc((void **)&d_info, sizeof(int));

    hipsolverEigType_t itype = HIPSOLVER_EIG_TYPE_1;
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;

    hipsolverDnDsygvd_bufferSize(cusolverH, itype, jobz, uplo, n, d_a, n, d_b, n, d_eig, &lwork);
    hipMalloc((void **)&work, sizeof(double) * lwork);

    hipsolverDnDsygvd(cusolverH, itype, jobz, uplo, n, d_a, n, d_b, n, d_eig, work, lwork, d_info);

    hipFree(work);
    hipFree(d_info);
}

int main() {
    double *d_a, *d_b, *d_eig;

    int dim[] = {500, 1000, 2000, 4000, 6000, 8000, 10000, 12000, 14000, 16000};
    int len = sizeof(dim) / sizeof(int);
    int trial = 5, execute = 20;
    float elapsed, mean, stddev;
    float times[execute];

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipsolverHandle_t cusolverH;
    hipsolverDnCreate(&cusolverH);

    printf("|   N   |   GPU (ms)  |  σ GPU (ms) |\n");
    printf("+-------+------------+-------------+\n");

    for (int index = 0; index < len; index++) {
        int n = dim[index];

        hipMalloc((void **)&d_a, n * n * sizeof(double));
        hipMalloc((void **)&d_b, n * n * sizeof(double));
        hipMalloc((void **)&d_eig, n * sizeof(double));

        for (int i = 0; i < trial + execute; i++) {
            gen_matrix(d_a, d_b, n);

            if (i >= trial) { // Только для execute замеров считаем время
                hipDeviceSynchronize();
                hipEventRecord(start, 0);

                eig_gpu(cusolverH, d_a, d_b, d_eig, n);

                hipEventRecord(stop, 0);
                hipEventSynchronize(stop);
                hipEventElapsedTime(&elapsed, start, stop);

                times[i - trial] = elapsed;
            } else {
                eig_gpu(cusolverH, d_a, d_b, d_eig, n); // Прогрев
            }
        }

        // Считаем среднее и СКО
        mean = 0.0f;
        for (int i = 0; i < execute; i++) mean += times[i];
        mean /= execute;

        stddev = 0.0f;
        for (int i = 0; i < execute; i++) stddev += (times[i] - mean) * (times[i] - mean);
        stddev = sqrtf(stddev / execute);

        printf("| %5d | %10.4f | %11.4f |\n", n, mean, stddev);

        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_eig);
    }

    hipsolverDnDestroy(cusolverH);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
