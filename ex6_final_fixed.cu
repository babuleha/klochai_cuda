#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <math.h>
#include <time.h>

#define THREADS_PER_BLOCK 32

// Генерация случайной матрицы (на GPU)
void random_doubles(double* d_a, int size, hiprandGenerator_t gen) {
    hiprandGenerateUniformDouble(gen, d_a, size);
}

// Генерация положительно определённой матрицы B = Aᵗ·A
void gen_dpm(hipblasHandle_t cublasH, double *d_out, double *d_tmp, int n) {
    const double alpha = 1.0, beta = 0.0;
    hipblasDgemm(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, n, &alpha, d_tmp, n, d_tmp, n, &beta, d_out, n);
}

// Генерация матриц A и B
void gen_matrix(hipblasHandle_t cublasH, hiprandGenerator_t gen, double *d_a, double *d_b, double *d_tmp, int n) {
    random_doubles(d_a, n * n, gen); // A — просто случайная
    random_doubles(d_tmp, n * n, gen);
    gen_dpm(cublasH, d_b, d_tmp, n); // B = tmpᵗ * tmp — положительно определённая
}

// Диагонализация на GPU
void eig_gpu(hipsolverHandle_t solverH, double *d_a, double *d_b, double *d_eig, int n) {
    int *d_info;
    double *work;
    int lwork;

    hipMalloc(&d_info, sizeof(int));
    hipsolverDnDsygvd_bufferSize(solverH, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR,
                                HIPBLAS_FILL_MODE_UPPER, n, d_a, n, d_b, n, d_eig, &lwork);

    hipMalloc(&work, lwork * sizeof(double));

    hipsolverDnDsygvd(solverH, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR,
                     HIPBLAS_FILL_MODE_UPPER, n, d_a, n, d_b, n, d_eig, work, lwork, d_info);

    hipFree(work);
    hipFree(d_info);
}

int main() {
    int dim[] = {500, 1000, 2000, 4000, 6000, 8000, 10000, 12000, 14000, 16000};
    int len = sizeof(dim) / sizeof(int);
    int trial = 5, execute = 20;
    float times[execute];
    float mean, stddev, elapsed;

    double *d_a, *d_b, *d_eig, *d_tmp;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipblasHandle_t cublasH;
    hipsolverHandle_t solverH;
    hipblasCreate(&cublasH);
    hipsolverDnCreate(&solverH);

    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, (unsigned long long)time(NULL));

    printf("|   N   |   GPU (ms)  |  σ GPU (ms) |\n");
    printf("+-------+------------+-------------+\n");

    for (int index = 0; index < len; index++) {
        int n = dim[index];
        hipMalloc(&d_a, n * n * sizeof(double));
        hipMalloc(&d_b, n * n * sizeof(double));
        hipMalloc(&d_tmp, n * n * sizeof(double));
        hipMalloc(&d_eig, n * sizeof(double));

        for (int i = 0; i < trial + execute; i++) {
            gen_matrix(cublasH, gen, d_a, d_b, d_tmp, n);

            if (i >= trial) {
                hipDeviceSynchronize();
                hipEventRecord(start, 0);

                eig_gpu(solverH, d_a, d_b, d_eig, n);

                hipEventRecord(stop, 0);
                hipEventSynchronize(stop);
                hipEventElapsedTime(&elapsed, start, stop);

                times[i - trial] = elapsed;
            } else {
                eig_gpu(solverH, d_a, d_b, d_eig, n); // прогрев
            }
        }

        mean = 0.0f;
        for (int i = 0; i < execute; i++) mean += times[i];
        mean /= execute;

        stddev = 0.0f;
        for (int i = 0; i < execute; i++) stddev += (times[i] - mean) * (times[i] - mean);
        stddev = sqrtf(stddev / execute);

        printf("| %5d | %10.4f | %11.4f |\n", n, mean, stddev);

        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_tmp);
        hipFree(d_eig);
    }

    hiprandDestroyGenerator(gen);
    hipblasDestroy(cublasH);
    hipsolverDnDestroy(solverH);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
